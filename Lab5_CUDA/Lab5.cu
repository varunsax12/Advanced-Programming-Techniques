#include "hip/hip_runtime.h"
/*
Description:
    Solution file for Lab 5.
    Implemented the solution to the heat plate problem using CUDA.
    1. The code initializes an input array for the initial temperature of the hot plate.
    2. It uses the averaging method to find the temperature of each element based on the
    temperature of its neighbours.
    3. Then it uses the jacobi iteration to assign the new temperature values back to the
    hot plate array.

    Top level reference for code: (as provided by Instructors on Piazza in Lab 5 Meta Thread @797, Author: Kuo-Wei-Lai)
    Piazza Link: https://piazza.com/class/ks93yix32l4s0?cid=797
    Code Reference: https://github.com/NVIDIA/cuda-samples/blob/master/Samples/vectorAdd/vectorAdd.cu
*/

#include <stdio.h>
#include <math.h>
#include <string.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

/*
* Function to check if command line option provided
* @param argc -> number of command line args
* @param argv -> array of command line args chars
* @param inputToCheck -> input argument to find
* @return int -> index of command line argument
*/
int checkCommandLineOption(int argc, char** argv, const char inputToCheck[])
{
    for (int i = 1; i < argc; ++i)
    {
        // compare strings
        if (strlen(argv[i]) != strlen(inputToCheck))
        {
            continue;
        }
        bool isAllEqual = true;
        for (int j = 0; j < strlen(inputToCheck); ++j)
        {
            if (argv[i][j] != inputToCheck[j])
            {
                isAllEqual = false;
                break;
            }
        }
        if (isAllEqual == true)
        {
            return i;
        }
    }
    return -1;
}

/*
* Function to print the error message for invalid command line inputs
*/
void printCommandLineError()
{
    printf("Invalid parameter, please check your values.\n");
}

/*
* Function to check for errors returns by the CUDA APIs
* @param errCode -> error code
* @param line -> line in which the error occured
*
* Reference for below code: hip/hip_runtime_api.h in cuda-samples-master.zip provided by Instructors in Canvas
*/
void checkCudaError(hipError_t code, int line)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "checkCudaError: %s %d\n", hipGetErrorString(code), line);
        exit(code);
    }
}

/*
* CUDA Kernel Device code
*
* Computes the new temperature of each index in the row-column matrix using
* the averaging method by taking the average of its 4 neighbours.
* @param outTemp -> pointer to the output temperature array
* @param inTemp -> pointer to the input temperature array
* @param dataSize -> size of each row of data
*/
__global__ void vector_update_temp(double* outTemp, const double* inTemp, const int dataSize)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int row = tid / dataSize;
    int col = tid % dataSize;
    // check if current element to check is either of th edge
    if (row >= dataSize || col >= dataSize)
    {
        return;
    }
    if (row == 0 || row == dataSize - 1 || col == 0 || col == dataSize - 1)
    {
        // copy forward the same temperature
        outTemp[tid] = inTemp[tid];
    }
    else
    {
        // if tid is not on any edge, then do below
        int top = (row - 1) * dataSize + col;
        int bottom = (row + 1) * dataSize + col;
        int left = row * dataSize + col - 1;
        int right = row * dataSize + col + 1;

        outTemp[tid] = (inTemp[top] + inTemp[bottom] + inTemp[left] + inTemp[right]) / 4;
    }
}

/*
* Main function to be run on the host machine
*/
int main(int argc, char** argv)
{
    int numInnerPoints = 0, numIter = 0, argIndex = 0;
    argIndex = checkCommandLineOption(argc, argv, "-q");
    if (argIndex != -1)
    {
        return 0;
    }
    argIndex = checkCommandLineOption(argc, argv, "-N");
    if (argIndex != -1)
    {
        numInnerPoints = atof(argv[argIndex + 1]);
        if (numInnerPoints == 0)
        {
            printCommandLineError();
            return 1;
        }
    }
    else
    {
        printCommandLineError();
        return 1;
    }
    argIndex = checkCommandLineOption(argc, argv, "-I");
    if (argIndex != -1)
    {
        numIter = atof(argv[argIndex + 1]);
        if (numInnerPoints == 0)
        {
            printCommandLineError();
            return 1;
        }
    }
    else
    {
        printCommandLineError();
        return 1;
    }

    // Variable to hold the dataSize based on number of inner points (+2 edges)
    int dataSize = numInnerPoints + 2;
    // Variable to hold the size to allocate memory for all arrays required
    size_t size = (dataSize * dataSize) * sizeof(double);

    // Pointers for the host machine temperature arrays, in and out
    double* h_inTemp = (double*)malloc(size);
    double* h_outTemp = (double*)malloc(size);

    // Verify if the array init was successful
    if (h_inTemp == NULL || h_outTemp == NULL)
    {
        fprintf(stderr, "Unable to allocate host array.\n");
        exit(EXIT_FAILURE);
    }

    // Init temp values
    for (int i = 0; i < dataSize; ++i)
    {
        for (int j = 0; j < dataSize; ++j)
        {
            int dataIndex = i * dataSize + j;
            h_inTemp[dataIndex] = 20.0;
            // check if it is in the 40 % of top row
            if (i == 0 && j > 0.3 * (dataSize - 1) && j < 0.7 * (dataSize - 1))
            {
                h_inTemp[dataIndex] = 100.0;
            }
        }
    }

    // Init the data on the device for input temp
    double* d_inTemp = NULL;
    checkCudaError(hipMalloc((void**)&d_inTemp, size), __LINE__);

    // Init the data on the device for output temp
    double* d_outTemp = NULL;
    checkCudaError(hipMalloc((void**)&d_outTemp, size), __LINE__);

    // Copy the host input temp to the device input temp
    checkCudaError(hipMemcpy(d_inTemp, h_inTemp, size, hipMemcpyHostToDevice), __LINE__);

    // Calculate the threads and blocks needed to launch CUDA kernel
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int threadsPerBlock = prop.maxThreadsPerBlock;
    int blocksPerGrid = (dataSize * dataSize + threadsPerBlock - 1) / threadsPerBlock;

    // Creating cuda events to use to time profiling
    // References from matrixMul.cu example in cuda-samples-master.zip
    // provided by Instructors in the modules section of Canvas
    hipEvent_t startEvent, stopEvent;
    checkCudaError(hipEventCreate(&startEvent), __LINE__);
    checkCudaError(hipEventCreate(&stopEvent), __LINE__);

    // Record the start event before starting kernel operations
    checkCudaError(hipEventRecord(startEvent, NULL), __LINE__);

    // Iterate over the temperature resolution logic for the required number of times
    for (int i = 0; i < numIter; ++i)
    {
        // Run the temperature update function on the kernel
        vector_update_temp << <blocksPerGrid, threadsPerBlock >> > (d_outTemp, d_inTemp, dataSize);
        // Check for erros
        checkCudaError(hipGetLastError(), __LINE__);
        // Copy the in output temperature into the input temperature for jacobi iteration
        checkCudaError(hipMemcpy(d_inTemp, d_outTemp, size, hipMemcpyDeviceToDevice), __LINE__);
    }

    // Record the stop event
    checkCudaError(hipEventRecord(stopEvent, NULL), __LINE__);

    // Wait for the stop event to complete
    checkCudaError(hipEventSynchronize(stopEvent), __LINE__);

    float timeElapsed = 0.0f;
    checkCudaError(hipEventElapsedTime(&timeElapsed, startEvent, stopEvent), __LINE__);

    // Copy the device output temp data into the host output temp
    checkCudaError(hipMemcpy(h_outTemp, d_outTemp, size, hipMemcpyDeviceToHost), __LINE__);


    // Write data into file
    FILE* filePointer;
    filePointer = fopen("finalTemperatures.csv", "w");

    // Print output temp
    for (int i = 0; i < dataSize; ++i)
    {
        for (int j = 0; j < dataSize; ++j)
        {
            fprintf(filePointer, "%0.6lf,", h_outTemp[i * (int)dataSize + j]);
        }
        fprintf(filePointer, "\n");
    }
    fclose(filePointer);

    // Free the device data arrays
    checkCudaError(hipFree(d_inTemp), __LINE__);
    checkCudaError(hipFree(d_outTemp), __LINE__);

    // Free the host data arrays
    free(h_inTemp);
    free(h_outTemp);

    // Print the time elapsed and exit
    printf("%0.2f\n", timeElapsed);

    return 0;
}
